#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <sys/time.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
	if (code != hipSuccess) {
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define THREADS 	256

// Knapsack parameters 
#define N   100 
#define W   500000


void CudaTimerStart(hipEvent_t* startGPU, hipEvent_t* stopGPU) {
	// Create the cuda events
	hipEventCreate(startGPU);
	hipEventCreate(stopGPU);
	// Record event on the default stream
	hipEventRecord(*startGPU, 0);
}

void CudaTimerStop(hipEvent_t* startGPU, hipEvent_t *stopGPU) {
    // Stop and destroy the timer
    float elapsed_gpu = 0.0;
	hipEventRecord(*stopGPU,0);
	hipEventSynchronize(*stopGPU);
	hipEventElapsedTime(&elapsed_gpu, *startGPU, *stopGPU);
	hipEventDestroy(*startGPU);
    hipEventDestroy(*stopGPU);
    printf("\nGPU time: %f (msec)\n", elapsed_gpu);
}

void initializeZerosFirstRow(float *arr) {
    for(int i =0; i<W; i++) {
        arr[i]= 0.0;
    }
}

void initializeValues(float *arr, int seed) {
	int i;
	srand(seed);
	for (i = 0; i < N; i++) {
        arr[i] = (float) (rand()%1000);
	}
}

void initializeWeights(int *arr, int seed) {
	int i;	
	srand(seed);
	for (i = 0; i < N; i++) {        
        arr[i] = (int) (rand()%20000);;
	}
}

void hostKnapsack(int *w, float* v, float *m, int *chosen) {
    int i, j;
    float with = 0, without = 0;

    for (i = 1; i < N; i++) {
        for (j = 1; j < W; j++) {
            if(j < w[i-1]) {
                // Skip
                m[i*W + j] = m[(i-1)*W + j]; 
                chosen[i*W + j] = 0;
            } else {
                // Should I take it or not
                without = m[(i-1)*W+j];
                with = m[(i-1)*W+(j-w[i-1])]+v[i-1];
                if(without >= with) {
                    m[i*W + j] = without;
                    chosen[i*W + j] = 0;
                } else {
                    m[i*W + j] = with;
                    chosen[i*W+j] = 1;
                }
            }
        }
    }
}

__global__ void GPU_knap(
    const int i,
    const int curv,
    const int curw,
    const int*__restrict__ DP_old,
    int* __restrict__ DP_new,
    int* __restrict__ Path,
    const int capacity) {

    const int offset = threadIdx.x+blockIdx.x*blockDim.x;
    const int which_warp = threadIdx.x>>5;
    const int tid_in_warp = threadIdx.x%32;

    __shared__ int s_cache[9][33];//do not really need to use __shared__ memory for this but did it anyway
    __shared__ int s_or[9][33];//ditto

    if(offset >= capacity) {
        return;
    }

    const int v1 = (offset >= curw) ? (DP_old[(offset-curw)] + curv) : -1;
    const int v0 = DP_old[offset];

    s_cache[which_warp][tid_in_warp] = (v1>=0 && v1>v0) ? v1 : v0;
    s_or[which_warp][tid_in_warp] = (v1>=0 && v1>v0) ? 1 : 0;
    __syncthreads();

    atomicExch(&DP_new[offset],s_cache[which_warp][tid_in_warp]);
    __syncthreads();

    if(s_or[which_warp][tid_in_warp] > 0) {
        atomicOr(&Path[offset],1);
    }
}

int main(int argc, char **argv) {
    // Select device
    CUDA_SAFE_CALL(hipSetDevice(0));

    // GPU Timing variables
    hipEvent_t startGPU, stopGPU;

    int dp_arr_size = N*W*sizeof(float);
    int chosen_arr_size = N*W*sizeof(int);
    int values_arr_size = N*sizeof(float);

    // Arrays on GPU global memory
    float *device_values, *device_DP;
	int *device_weights, *device_chosen;
    CUDA_SAFE_CALL(hipMalloc((void **)&device_weights, values_arr_size));
	CUDA_SAFE_CALL(hipMalloc((void **)&device_values, values_arr_size));
    CUDA_SAFE_CALL(hipMalloc((void **)&device_DP, dp_arr_size));
    CUDA_SAFE_CALL(hipMalloc((void **)&device_chosen, chosen_arr_size));
    
    // 2D arrays on host memory
    float *host_values, *host_DP;
    int *host_weights, *host_chosen;
    host_weights = (int *) malloc(values_arr_size);
    host_chosen = (int *)malloc(chosen_arr_size);
	host_values = (float *) malloc(values_arr_size);
	host_DP = (float *)malloc(dp_arr_size);
    
    // Initialize the arrays on CPU
    initializeValues(host_values, 1251);
    initializeWeights(host_weights, 1251);
    initializeZerosFirstRow(host_DP); // Marks the entire first row as zeros
    
    // Transfer the 2d-arrays to the GPU memory
	CUDA_SAFE_CALL(hipMemcpy(device_values, host_values, N*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(device_weights, host_weights, N*sizeof(int), hipMemcpyHostToDevice));

    CudaTimerStart(&startGPU, &stopGPU);

	dim3 dimGrid(((W+THREADS-1)/THREADS),1,1);
    dim3 dimBlock((THREADS, 1, 1);

    for(; ii<N; ii++) {
        GPU_knap<<<dimGrid,dimBlock>>>(
            ii,
            device_values[ii-1],
            device_weights[ii-1],
            (int*)(&d_DP[(ii-1)*W]),
            (int*)(&d_DP[ii*W]),
            (int*)(&d_Path[ii*W]),
            W);       
        CUDA_SAFE_CALL(hipDeviceSynchronize());
    }

	// Check for errors during launch
    CUDA_SAFE_CALL(hipPeekAtLastError());
    float *device_result = malloc(N*W*sizeof(float));
    CUDA_SAFE_CALL(hipMemcpy(device_result, device_DP, N*W*sizeof(float)));
    printf("GPU Result %f\n", device_result[(N*W) + W - 1]);    

    // Transfer the results back to the host
    //CUDA_SAFE_CALL(hipMemcpy(host_deviceResCopy, device_res, allocSize2D, hipMemcpyDeviceToHost));
    CudaTimerStop(&startGPU, &stopGPU);

    // **************** CPU BASELINE **************************************
    // Calculate time
    struct timeval t1, t2;
    gettimeofday(&t1, 0);
    // Compute on CPU
    hostKnapsack(host_weights, host_values, host_DP, host_chosen);
    gettimeofday(&t2, 0);
    double total_cpu_time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    printf("Time to generate:  %3.1f ms \n", total_cpu_time);


    printf("Result %f\n", host_DP[(N*W) + W - 1]);

	// Free-up device and host memory
    CUDA_SAFE_CALL(hipFree(device_weights));
    CUDA_SAFE_CALL(hipFree(device_values));
    CUDA_SAFE_CALL(hipFree(device_chosen));
    CUDA_SAFE_CALL(hipFree(device_DP));
    free(host_weights);
    free(host_values);
    free(host_chosen);
    free(host_DP);
	return 0;
}
